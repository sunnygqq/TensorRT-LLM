#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdexcept>
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "hipcub/hipcub.hpp"
#endif

#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/common/stringUtils.h"
#include "tensorrt_llm/kernels/samplingTopKKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

template <typename T>
__global__ void addBiasEndMask(T* logits, const T* bias, const int* endIds, const FinishedState* finished,
    const int vocabSize, const int vocabSizePadded)
{
    int bid = blockIdx.x;
    const FinishedState finishState = finished != nullptr ? finished[bid] : FinishedState::empty();
    if (finishState.isSkipDecoding())
    {
        return;
    }

    bool finish = finishState.isFinished();
    int offset = bid * vocabSizePadded;

    const bool IS_FP16 = std::is_same<T, half>::value;
    const T MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;
    for (int tid = threadIdx.x; tid < vocabSizePadded; tid += blockDim.x)
    {
        if (tid >= vocabSize)
        {
            logits[offset + tid] = -MAX_T_VAL;
        }
        else if (finish)
        {
            logits[offset + tid] = (tid == endIds[bid]) ? MAX_T_VAL : -MAX_T_VAL;
        }
        else
        {
            if (bias != nullptr)
            {
                logits[offset + tid] += bias[tid];
            }
        }
    }
}

template <typename T>
void invokeAddBiasEndMask(T* logits, const T* bias, const int* endIds, const FinishedState* finished,
    const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream)
{
    dim3 grid(batchSize);
    dim3 block(min(vocabSizePadded, 1024));
    // n is the vocabSize, e.g., 30000, 7000.... vocabSize is usually very big.
    addBiasEndMask<<<grid, block, 0, stream>>>(logits, bias, endIds, finished, vocabSize, vocabSizePadded);
}

template void invokeAddBiasEndMask(float* logits, const float* bias, const int* endIds, const FinishedState* finished,
    const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream);

template void invokeAddBiasEndMask(half* logits, const half* bias, const int* endIds, const FinishedState* finished,
    const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream);

template <typename T, int BLOCK_SIZE_, int BLOCKS_PER_BEAM_>
__global__ void topKStage1(const T* __restrict logProbs, T* tmpLogProbs, int* topKTmpIdBuf, T* topKTmpValBuf,
    const FinishedState* finished, const int maxTopK, const int* topKs, const int vocabSize, const int* endIds,
    const bool* skipDecode)
{
    typedef hipcub::BlockReduce<TopK_2<T>, BLOCK_SIZE_> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int batchId = bid / BLOCKS_PER_BEAM_; // row id for logProbs
    const FinishedState finishState = finished != nullptr ? finished[batchId] : FinishedState::empty();
    if ((skipDecode != nullptr && skipDecode[batchId]) || (finishState.isSkipDecoding()))
    {
        return;
    }
    const int blockLane = bid % BLOCKS_PER_BEAM_;                // block id for a beam
    const int k = (topKs != nullptr) ? topKs[batchId] : maxTopK; // batchId = batch index

    const int tmpLogBufIndex = batchId * vocabSize;
    const int tmpTopKBufIndex = batchId * BLOCKS_PER_BEAM_ * maxTopK + blockLane * k;

    TopK_2<T> partial;
    const bool IS_FP16 = std::is_same<T, half>::value;
    const T MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    if (finished != nullptr && finishState.isFinished())
    {
        if (tid < k)
        {
            const int index = tmpTopKBufIndex + tid;
            if (blockLane == 0 && tid == 0)
            {
                const int endId = endIds[batchId];
                topKTmpIdBuf[index] = tmpLogBufIndex + endId;
                topKTmpValBuf[index] = logProbs[tmpLogBufIndex + endId];
            }
            else
            {
                topKTmpIdBuf[index] = -1;
                topKTmpValBuf[index] = -MAX_T_VAL;
            }
        }
        return;
    }

    for (int elemId = tid + blockLane * BLOCK_SIZE_; elemId < vocabSize; elemId += BLOCK_SIZE_ * BLOCKS_PER_BEAM_)
    {
        int index = elemId + tmpLogBufIndex;
        tmpLogProbs[index] = logProbs[index];
    }

    for (int ite = 0; ite < k; ite++)
    {
        partial.init();
#pragma unroll
        for (int elemId = tid + blockLane * BLOCK_SIZE_; elemId < vocabSize; elemId += BLOCK_SIZE_ * BLOCKS_PER_BEAM_)
        {
            int index = elemId + tmpLogBufIndex;
            partial.insert(tmpLogProbs[index], index);
        }

        TopK_2<T> total = BlockReduce(tempStorage).Reduce(partial, reduce_topk_op_2<T>);

        if (tid == 0)
        {
            const int index = tmpTopKBufIndex + ite;
            topKTmpIdBuf[index] = total.p;
            topKTmpValBuf[index] = total.u;
            if (total.p >= 0)
            {
                tmpLogProbs[total.p] = -MAX_T_VAL;
            }
        }
        __syncthreads();
    }
}

template <typename T, int BLOCK_SIZE_, int BLOCKS_PER_BEAM_>
__global__ void topKStage2Sampling(const int* __restrict topKTmpIdBuf, T* topKTmpValBuf, int** ids,
    int* sequenceLengths, const FinishedState* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, const int maxTopK, const int* topKs, const float topP, const float* topPs,
    hiprandState_t* curandstate, const int* endIds, const int vocabSize, const bool* skipDecode)
{
    const bool IS_FP16 = std::is_same<T, half>::value;
    const T MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    const int tid = threadIdx.x;
    const int batchId = blockIdx.x;
    const FinishedState finishState = finishedInput != nullptr ? finishedInput[batchId] : FinishedState::empty();
    if ((skipDecode != nullptr && skipDecode[batchId]) || (finishState.isSkipDecoding()))
    {
        return;
    }

    const int k = (topKs != nullptr) ? topKs[batchId] : maxTopK;
    const float probThreshold = (topPs != nullptr) ? topPs[batchId] : topP;
    const int size = k * BLOCKS_PER_BEAM_;
    const int stride = maxTopK * BLOCKS_PER_BEAM_;

    typedef hipcub::BlockReduce<TopK_2<float>, BLOCK_SIZE_> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;
    extern __shared__ char array[];
    __shared__ float s_sum;
    T* s_val = topKTmpValBuf + batchId * stride;
    int* s_id = reinterpret_cast<int*>(array);
    if (tid == 0)
    {
        s_sum = 0.0f;
    }
    TopK_2<float> partial;

    if (finishState.isFinished())
    {
        if (finishedOutput != nullptr)
        {
            finishedOutput[batchId] = finishState;
        }
        return;
    }

    float* s_val2 = reinterpret_cast<float*>(s_id + k);
    float maxLogit;
    for (int ite = 0; ite < k; ite++)
    {
        partial.init();
#pragma unroll
        for (int i = tid; i < size; i += BLOCK_SIZE_)
        {
            partial.insert((float) s_val[i], i);
        }

        TopK_2<float> total = BlockReduce(tempStorage).Reduce(partial, reduce_topk_op_2<float>);

        if (tid == 0)
        {
            if (ite == 0)
            {
                maxLogit = total.u;
            }
            s_id[ite] = total.p;
            s_val[total.p] = -MAX_T_VAL;

            // when cumLogProbs are computed, topKTmpValBuf (logits_buf_) are
            // already pre-processed by softmax_kernel
            if (cumLogProbs == nullptr && outputLogProbs == nullptr)
            {
                total.u = __expf(total.u - maxLogit);
            }
            s_val2[ite] = total.u;
            s_sum += total.u;
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        float randNum = (float) hiprand_uniform(curandstate + blockIdx.x) * probThreshold * s_sum;
        for (int i = 0; i < k; i++)
        {
            float expLogit = s_val2[i];
            randNum = randNum - expLogit;
            if (randNum <= 0.0f || i == k - 1)
            {
                int idx = s_id[i];
                // If s_id is -1 here we force output token to the last from vocabulary to get vivid indicator of smth
                // going wrong for the debug
                auto outputId = idx != -1 ? topKTmpIdBuf[batchId * stride + idx] % vocabSize : vocabSize - 1;
                ids[batchId][sequenceLengths[batchId]] = outputId;
                if (cumLogProbs != nullptr || outputLogProbs != nullptr)
                {
                    float logProb = logf(expLogit);
                    if (cumLogProbs != nullptr)
                    {
                        cumLogProbs[batchId] += logProb;
                    }
                    if (outputLogProbs != nullptr)
                    {
                        // 'outputLogProbs' is the probability induced by the top-k
                        // sampling. We normalize the probability 'expLogit' of the
                        // selected token by the probability 's_sum' of a set of top-k
                        // tokens, meaning the logProb is the probability of the selected
                        // token, conditioned on the event that it is selected, i.e.,
                        //   log_prob = log P(i | i is in top-k) = log(expLogit / s_sum).
                        outputLogProbs[batchId] = logProb - logf(s_sum);
                    }
                }
                break;
            }
        }
        if (sequenceLengths != nullptr && finishedOutput != nullptr)
        {
            const int seqLen = sequenceLengths[batchId];
            if (ids[batchId][seqLen] == endIds[batchId])
            {
                finishedOutput[batchId].setFinishedEOS();
                // Do not increase seq len when EOS is generated. Seq len should always contain only tokens to be
                // outputted
            }
            else
            {
                // We don't need to set output finished state as it is assumed to be in non finished state
                sequenceLengths[batchId] += 1;
            }
        }
    }
}

#define CASE_K(K_MAX, BLOCK_SIZE_1_, BLOCK_SIZE_2_, BLOCKS_PER_BEAM_)                                                  \
    topKStage1<T, BLOCK_SIZE_1_, BLOCKS_PER_BEAM_>                                                                     \
        <<<batchSize * BLOCKS_PER_BEAM_, BLOCK_SIZE_1_, 0, stream>>>(logProbs, tempLogProbs, topKTmpIdBuf,             \
            topKTmpValBuf, finishedInput, maxTopK, topKs, vocabSize, endIds, skipDecode);                              \
    topKStage2Sampling<T, BLOCK_SIZE_2_, BLOCKS_PER_BEAM_>                                                             \
        <<<batchSize, BLOCK_SIZE_2_, K_MAX * sizeof(int) + K_MAX * sizeof(float), stream>>>(topKTmpIdBuf,              \
            topKTmpValBuf, ids, sequenceLengths, finishedInput, finishedOutput, cumLogProbs, outputLogProbs, maxTopK,  \
            topKs, topP, topPs, curandstate, endIds, vocabSize, skipDecode);                                           \
    break;

template <typename T>
void invokeBatchTopKSampling(void* workspace, size_t& workspaceSize, const T* logProbs, int** ids, int* sequenceLengths,
    const FinishedState* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    hiprandState_t* curandstate, const int maxTopK, const int* topKs, const float topP, const float* topPs,
    const int vocabSizePadded, const int* endIds, hipStream_t stream, const int batchSize, const bool* skipDecode)
{
    TLLM_LOG_DEBUG("%s start", __PRETTY_FUNCTION__);

    // Not allow an ambiguous inputs topP and topPs.
    assert(topP == 1.0f || topPs == nullptr);
    const int vocabSize = vocabSizePadded;
    const int maxBlockPerBeam = 8;
    int tempLogProbsBufSize = batchSize * vocabSize;                // type float
    int topKTmpIdsBufSize = batchSize * maxTopK * maxBlockPerBeam;  // type int
    int topKTmpValBuf_size = batchSize * maxTopK * maxBlockPerBeam; // type float

    // prevent memory misaligned address
    tempLogProbsBufSize = (int) (ceil(tempLogProbsBufSize / 4.)) * 4;
    topKTmpIdsBufSize = (int) (ceil(topKTmpIdsBufSize / 4.)) * 4;
    topKTmpValBuf_size = (int) (ceil(topKTmpValBuf_size / 4.)) * 4;

    if (workspace == nullptr)
    {
        workspaceSize
            = sizeof(T) * tempLogProbsBufSize + sizeof(int) * topKTmpIdsBufSize + sizeof(T) * topKTmpValBuf_size;
        return;
    }

    T* tempLogProbs = (T*) workspace;
    int* topKTmpIdBuf = (int*) (tempLogProbs + tempLogProbsBufSize);
    T* topKTmpValBuf = (T*) (topKTmpIdBuf + topKTmpIdsBufSize);

    int logMaxTopK(0);
    int recursor(maxTopK - 1);
    while (recursor >>= 1)
        ++logMaxTopK;
    switch (logMaxTopK)
    {
    case 0:
    case 1:
    case 2:
    case 3: // 0 < maxTopK <= 16
        CASE_K(16, 128, 128, 8);
    case 4: // 16 < maxTopK <= 32
        CASE_K(32, 256, 128, 8);
    case 5: // 32 < maxTopK <= 64
        CASE_K(64, 256, 256, 8);
    case 6:
    case 7:
    case 8:
    case 9: // 64 < maxTopK <= 1024
        CASE_K(1024, 256, 256, 8);
    default: throw std::domain_error(fmtstr("top-k kernel supports 1<=k<=1024 but got k=%d", maxTopK));
    }
}

#undef CASE_K

template void invokeBatchTopKSampling(void* workspace, size_t& workspaceSize, const float* logProbs, int** ids,
    int* sequenceLengths, const FinishedState* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, hiprandState_t* curandstate, const int maxTopK, const int* topKs, const float topP,
    const float* topPs, const int vocabSizePadded, const int* endIds, hipStream_t stream, const int batchSize,
    const bool* skipDecode);

template void invokeBatchTopKSampling(void* workspace, size_t& workspaceSize, const half* logProbs, int** ids,
    int* sequenceLengths, const FinishedState* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, hiprandState_t* curandstate, const int maxTopK, const int* topKs, const float topP,
    const float* topPs, const int vocabSizePadded, const int* endIds, hipStream_t stream, const int batchSize,
    const bool* skipDecode);

template <typename T>
void invokeTopKSampling(void* workspace, size_t& workspaceSize, const T* logProbs, int** ids, int* sequenceLengths,
    const FinishedState* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    hiprandState_t* curandstate, const int topK, const float topP, const int vocabSizePadded, const int* endIds,
    hipStream_t stream, const int batchSize, const bool* skipDecode)
{
    invokeBatchTopKSampling(workspace, workspaceSize, logProbs, ids, sequenceLengths, finishedInput, finishedOutput,
        cumLogProbs, outputLogProbs, curandstate, topK, nullptr, topP, nullptr, vocabSizePadded, endIds, stream,
        batchSize, skipDecode);
}

template void invokeTopKSampling(void* workspace, size_t& workspaceSize, const float* logProbs, int** ids,
    int* sequenceLengths, const FinishedState* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, hiprandState_t* curandstate, const int topK, const float topP, const int vocabSizePadded,
    const int* endIds, hipStream_t stream, const int batchSize, const bool* skipDecode);

template void invokeTopKSampling(void* workspace, size_t& workspaceSize, const half* logProbs, int** ids,
    int* sequenceLengths, const FinishedState* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, hiprandState_t* curandstate, const int topK, const float topP, const int vocabSizePadded,
    const int* endIds, hipStream_t stream, const int batchSize, const bool* skipDecode);

} // namespace kernels
} // namespace tensorrt_llm
